#include "hip/hip_runtime.h"
#include <sudoku/cuda/error_check.h>
#include <sudoku/cuda/guess_stack.cuh>
#include "guess_stack_kernels.h"

using namespace sudoku::cuda;

__global__ void guessStackPushKernel(CellCount* globalGuessStack, CellCount* globalGuessStackSize, CellCount cellPos)
{
    extern __shared__ CellCount sharedGuessStack[];
    GuessStack guessStack(globalGuessStack, globalGuessStackSize, sharedGuessStack);
    guessStack.push(cellPos);
}

__global__ void guessStackPopKernel(CellCount* globalGuessStack, CellCount* globalGuessStackSize, CellCount* outPos)
{
    extern __shared__ CellCount sharedGuessStack[];
    GuessStack guessStack(globalGuessStack, globalGuessStackSize, sharedGuessStack);
    CellCount myOutPos = guessStack.pop();
    if (threadIdx.x == 0) {
        *outPos = myOutPos;
    }
}

GuessStackKernels::GuessStackKernels(CellCount maxStackSize)
    : hostGuessStack_(maxStackSize, 0)
    , hostGuessStackSize_(1, 0)
    , deviceGuessStack_(hostGuessStack_)
    , deviceGuessStackSize_(hostGuessStackSize_)
    , threadCount_(maxStackSize)
    , sharedMemSize_(sizeof(CellCount) * maxStackSize)
{}

void GuessStackKernels::copyToHost()
{
    hostGuessStack_ = deviceGuessStack_.copyToHost();
    hostGuessStackSize_ = deviceGuessStackSize_.copyToHost();
}

void GuessStackKernels::push(CellCount cellPos)
{
    guessStackPushKernel<<<1, threadCount_, sharedMemSize_>>>(
        deviceGuessStack_.get(), deviceGuessStackSize_.get(), cellPos
    );
    ErrorCheck::lastError();
    copyToHost();
}

CellCount GuessStackKernels::pop()
{
    DeviceBuffer<CellCount> outPos(1);
    guessStackPopKernel<<<1, threadCount_, sharedMemSize_>>>(
        deviceGuessStack_.get(), deviceGuessStackSize_.get(), outPos.get()
    );
    ErrorCheck::lastError();
    copyToHost();
    return outPos.copyToHost()[0];
}
